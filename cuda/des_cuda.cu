/* This program is aimed to do the cuda implementation of DESnuts */

#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdint.h>
#include <inttypes.h>
#include <stdio.h>

/**********************************************************************/
/*                                                                    */
/*                            DES TABLES                              */
/*                                                                    */
/**********************************************************************/

/* The number of bytes need for storing all the DES TABLES are:
 * 64 * 4 + 64 * 4 + 56 * 4 + 48 * 4 + 48 * 4 + 32 * 4 + 8 * 64 * 4 
 * = 3296 bytes
 */

/*
 *  IP: Output bit table_DES_IP[i] equals input bit i.
 */
static int table_DES_IP[64] = {
    39,  7, 47, 15, 55, 23, 63, 31,
    38,  6, 46, 14, 54, 22, 62, 30,
    37,  5, 45, 13, 53, 21, 61, 29,
    36,  4, 44, 12, 52, 20, 60, 28,
    35,  3, 43, 11, 51, 19, 59, 27,
    34,  2, 42, 10, 50, 18, 58, 26,
    33,  1, 41,  9, 49, 17, 57, 25,
    32,  0, 40,  8, 48, 16, 56, 24
};


/*
 *  FP: Output bit table_DES_FP[i] equals input bit i.
 */
static int table_DES_FP[64] = {
    57, 49, 41, 33, 25, 17,  9,  1,
    59, 51, 43, 35, 27, 19, 11,  3,
    61, 53, 45, 37, 29, 21, 13,  5,
    63, 55, 47, 39, 31, 23, 15,  7,
    56, 48, 40, 32, 24, 16,  8,  0,
    58, 50, 42, 34, 26, 18, 10,  2,
    60, 52, 44, 36, 28, 20, 12,  4,
    62, 54, 46, 38, 30, 22, 14,  6
};


/*
 *  PC1: Permutation choice 1, used to pre-process the key
 */
static int table_DES_PC1[56] = {
    27, 19, 11, 31, 39, 47, 55,
    26, 18, 10, 30, 38, 46, 54,
    25, 17,  9, 29, 37, 45, 53,
    24, 16,  8, 28, 36, 44, 52,
    23, 15,  7,  3, 35, 43, 51,
    22, 14,  6,  2, 34, 42, 50,
    21, 13,  5,  1, 33, 41, 49,
    20, 12,  4,  0, 32, 40, 48
};


/*
 *  PC2: Map 56-bit round key to a 48-bit subkey
 */
static int table_DES_PC2[48] = {
    24, 27, 20,  6, 14, 10,  3, 22,
    0, 17,  7, 12,  8, 23, 11,  5,
    16, 26,  1,  9, 19, 25,  4, 15,
    54, 43, 36, 29, 49, 40, 48, 30,
    52, 44, 37, 33, 46, 35, 50, 41,
    28, 53, 51, 55, 32, 45, 39, 42
};


/*
 *  E: Expand 32-bit R to 48 bits.
 */
static int table_DES_E[48] = {
    31,  0,  1,  2,  3,  4,  3,  4,
    5,  6,  7,  8,  7,  8,  9, 10,
    11, 12, 11, 12, 13, 14, 15, 16,
    15, 16, 17, 18, 19, 20, 19, 20,
    21, 22, 23, 24, 23, 24, 25, 26,
    27, 28, 27, 28, 29, 30, 31,  0
};


/*
 *  P: Permutation of S table outputs
 */
static int table_DES_P[32] = {
    11, 17,  5, 27, 25, 10, 20,  0,
    13, 21,  3, 28, 29,  7, 18, 24,
    31, 22, 12,  6, 26,  2, 16,  8,
    14, 30,  4, 19,  1,  9, 15, 23
};


/*
 *  S Tables: Introduce nonlinearity and avalanche
 */
static int table_DES_S[8][64] = {
    /* table S[0] */
    {   13,  1,  2, 15,  8, 13,  4,  8,  6, 10, 15,  3, 11,  7,  1,  4,
        10, 12,  9,  5,  3,  6, 14, 11,  5,  0,  0, 14, 12,  9,  7,  2,
        7,  2, 11,  1,  4, 14,  1,  7,  9,  4, 12, 10, 14,  8,  2, 13,
        0, 15,  6, 12, 10,  9, 13,  0, 15,  3,  3,  5,  5,  6,  8, 11  },
    /* table S[1] */
    {    4, 13, 11,  0,  2, 11, 14,  7, 15,  4,  0,  9,  8,  1, 13, 10,
        3, 14, 12,  3,  9,  5,  7, 12,  5,  2, 10, 15,  6,  8,  1,  6,
        1,  6,  4, 11, 11, 13, 13,  8, 12,  1,  3,  4,  7, 10, 14,  7,
        10,  9, 15,  5,  6,  0,  8, 15,  0, 14,  5,  2,  9,  3,  2, 12  },
    /* table S[2] */
    {   12, 10,  1, 15, 10,  4, 15,  2,  9,  7,  2, 12,  6,  9,  8,  5,
        0,  6, 13,  1,  3, 13,  4, 14, 14,  0,  7, 11,  5,  3, 11,  8,
        9,  4, 14,  3, 15,  2,  5, 12,  2,  9,  8,  5, 12, 15,  3, 10,
        7, 11,  0, 14,  4,  1, 10,  7,  1,  6, 13,  0, 11,  8,  6, 13  },
    /* table S[3] */
    {    2, 14, 12, 11,  4,  2,  1, 12,  7,  4, 10,  7, 11, 13,  6,  1,
        8,  5,  5,  0,  3, 15, 15, 10, 13,  3,  0,  9, 14,  8,  9,  6,
        4, 11,  2,  8,  1, 12, 11,  7, 10,  1, 13, 14,  7,  2,  8, 13,
        15,  6,  9, 15, 12,  0,  5,  9,  6, 10,  3,  4,  0,  5, 14,  3  },
    /* table S[4] */
    {    7, 13, 13,  8, 14, 11,  3,  5,  0,  6,  6, 15,  9,  0, 10,  3,
        1,  4,  2,  7,  8,  2,  5, 12, 11,  1, 12, 10,  4, 14, 15,  9,
        10,  3,  6, 15,  9,  0,  0,  6, 12, 10, 11,  1,  7, 13, 13,  8,
        15,  9,  1,  4,  3,  5, 14, 11,  5, 12,  2,  7,  8,  2,  4, 14  },
    /* table S[5] */
    {   10, 13,  0,  7,  9,  0, 14,  9,  6,  3,  3,  4, 15,  6,  5, 10,
        1,  2, 13,  8, 12,  5,  7, 14, 11, 12,  4, 11,  2, 15,  8,  1,
        13,  1,  6, 10,  4, 13,  9,  0,  8,  6, 15,  9,  3,  8,  0,  7,
        11,  4,  1, 15,  2, 14, 12,  3,  5, 11, 10,  5, 14,  2,  7, 12  },
    /* table S[6] */
    {   15,  3,  1, 13,  8,  4, 14,  7,  6, 15, 11,  2,  3,  8,  4, 14,
        9, 12,  7,  0,  2,  1, 13, 10, 12,  6,  0,  9,  5, 11, 10,  5,
        0, 13, 14,  8,  7, 10, 11,  1, 10,  3,  4, 15, 13,  4,  1,  2,
        5, 11,  8,  6, 12,  7,  6, 12,  9,  0,  3,  5,  2, 14, 15,  9  },
    /* table S[7] */
    {   14,  0,  4, 15, 13,  7,  1,  4,  2, 14, 15,  2, 11, 13,  8,  1,
        3, 10, 10,  6,  6, 12, 12, 11,  5,  9,  9,  5,  0,  3,  7,  8,
        4, 15,  1, 12, 14,  8,  8,  2, 13,  4,  6,  9,  2,  1, 11,  7,
        15,  5, 12, 11,  9,  3,  7, 14,  3, 10, 10,  0,  5,  6,  0, 13  }
};

#define MASK56(n) ((n) & 0x00FFFFFFFFFFFFFF)

void print_bits_array(uint64_t n) {
    printf("%lX\n", n);
}

#define COMPUTE_ROUND_KEY(roundKey, key)        \
    roundKey |= ((key & ((1UL) << 0)) << (27));     \
    roundKey |= ((key & ((1UL) << 1)) << (18));     \
    roundKey |= ((key & ((1UL) << 2)) << (9));  \
    roundKey |= ((key & ((1UL) << 3)) << (28));     \
    roundKey |= ((key & ((1UL) << 4)) << (35));     \
    roundKey |= ((key & ((1UL) << 5)) << (42));     \
    roundKey |= ((key & ((1UL) << 6)) << (49));     \
    roundKey |= ((key & ((1UL) << 7)) << (19));     \
    roundKey |= ((key & ((1UL) << 8)) << (10));     \
    roundKey |= ((key & ((1UL) << 9)) << (1));  \
    roundKey |= ((key & ((1UL) << 10)) << (20));    \
    roundKey |= ((key & ((1UL) << 11)) << (27));    \
    roundKey |= ((key & ((1UL) << 12)) << (34));    \
    roundKey |= ((key & ((1UL) << 13)) << (41));    \
    roundKey |= ((key & ((1UL) << 14)) << (11));    \
    roundKey |= ((key & ((1UL) << 15)) << (2));     \
    roundKey |= ((key & ((1UL) << 16)) >> (7));     \
    roundKey |= ((key & ((1UL) << 17)) << (12));    \
    roundKey |= ((key & ((1UL) << 18)) << (19));    \
    roundKey |= ((key & ((1UL) << 19)) << (26));    \
    roundKey |= ((key & ((1UL) << 20)) << (33));    \
    roundKey |= ((key & ((1UL) << 21)) << (3));     \
    roundKey |= ((key & ((1UL) << 22)) >> (6));     \
    roundKey |= ((key & ((1UL) << 23)) >> (15));    \
    roundKey |= ((key & ((1UL) << 24)) << (4));     \
    roundKey |= ((key & ((1UL) << 25)) << (11));    \
    roundKey |= ((key & ((1UL) << 26)) << (18));    \
    roundKey |= ((key & ((1UL) << 27)) << (25));    \
    roundKey |= ((key & ((1UL) << 28)) >> (5));     \
    roundKey |= ((key & ((1UL) << 29)) >> (14));    \
    roundKey |= ((key & ((1UL) << 30)) >> (23));    \
    roundKey |= ((key & ((1UL) << 31)) >> (28));    \
    roundKey |= ((key & ((1UL) << 32)) << (3));     \
    roundKey |= ((key & ((1UL) << 33)) << (10));    \
    roundKey |= ((key & ((1UL) << 34)) << (17));    \
    roundKey |= ((key & ((1UL) << 35)) >> (13));    \
    roundKey |= ((key & ((1UL) << 36)) >> (22));    \
    roundKey |= ((key & ((1UL) << 37)) >> (31));    \
    roundKey |= ((key & ((1UL) << 38)) >> (36));    \
    roundKey |= ((key & ((1UL) << 39)) >> (5));     \
    roundKey |= ((key & ((1UL) << 40)) << (2));     \
    roundKey |= ((key & ((1UL) << 41)) << (9));     \
    roundKey |= ((key & ((1UL) << 42)) >> (21));    \
    roundKey |= ((key & ((1UL) << 43)) >> (30));    \
    roundKey |= ((key & ((1UL) << 44)) >> (39));    \
    roundKey |= ((key & ((1UL) << 45)) >> (44));    \
    roundKey |= ((key & ((1UL) << 46)) >> (13));    \
    roundKey |= ((key & ((1UL) << 47)) >> (6));     \
    roundKey |= ((key & ((1UL) << 48)) << (1));     \
    roundKey |= ((key & ((1UL) << 49)) >> (29));    \
    roundKey |= ((key & ((1UL) << 50)) >> (38));    \
    roundKey |= ((key & ((1UL) << 51)) >> (47));    \
    roundKey |= ((key & ((1UL) << 52)) >> (52));    \
    roundKey |= ((key & ((1UL) << 53)) >> (21));    \
    roundKey |= ((key & ((1UL) << 54)) >> (14));    \
    roundKey |= ((key & ((1UL) << 55)) >> (7));     \


#define COMPUTE_IP(L, R, in)            \
    uint64_t output = 0UL;                    \
    output |= ((in & ((1UL) << 63)) >> (39));   \
    output |= ((in & ((1UL) << 62)) >> (6));    \
    output |= ((in & ((1UL) << 61)) >> (45));   \
    output |= ((in & ((1UL) << 60)) >> (12));   \
    output |= ((in & ((1UL) << 59)) >> (51));   \
    output |= ((in & ((1UL) << 58)) >> (18));   \
    output |= ((in & ((1UL) << 57)) >> (57));   \
    output |= ((in & ((1UL) << 56)) >> (24));   \
    output |= ((in & ((1UL) << 55)) >> (30));   \
    output |= ((in & ((1UL) << 54)) << (3));    \
    output |= ((in & ((1UL) << 53)) >> (36));   \
    output |= ((in & ((1UL) << 52)) >> (3));    \
    output |= ((in & ((1UL) << 51)) >> (42));   \
    output |= ((in & ((1UL) << 50)) >> (9));    \
    output |= ((in & ((1UL) << 49)) >> (48));   \
    output |= ((in & ((1UL) << 48)) >> (15));   \
    output |= ((in & ((1UL) << 47)) >> (21));   \
    output |= ((in & ((1UL) << 46)) << (12));   \
    output |= ((in & ((1UL) << 45)) >> (27));   \
    output |= ((in & ((1UL) << 44)) << (6));    \
    output |= ((in & ((1UL) << 43)) >> (33));   \
    output |= ((in & ((1UL) << 42)) << (0));    \
    output |= ((in & ((1UL) << 41)) >> (39));   \
    output |= ((in & ((1UL) << 40)) >> (6));    \
    output |= ((in & ((1UL) << 39)) >> (12));   \
    output |= ((in & ((1UL) << 38)) << (21));   \
    output |= ((in & ((1UL) << 37)) >> (18));   \
    output |= ((in & ((1UL) << 36)) << (15));   \
    output |= ((in & ((1UL) << 35)) >> (24));   \
    output |= ((in & ((1UL) << 34)) << (9));    \
    output |= ((in & ((1UL) << 33)) >> (30));   \
    output |= ((in & ((1UL) << 32)) << (3));    \
    output |= ((in & ((1UL) << 31)) >> (3));    \
    output |= ((in & ((1UL) << 30)) << (30));   \
    output |= ((in & ((1UL) << 29)) >> (9));    \
    output |= ((in & ((1UL) << 28)) << (24));   \
    output |= ((in & ((1UL) << 27)) >> (15));   \
    output |= ((in & ((1UL) << 26)) << (18));   \
    output |= ((in & ((1UL) << 25)) >> (21));   \
    output |= ((in & ((1UL) << 24)) << (12));   \
    output |= ((in & ((1UL) << 23)) << (6));    \
    output |= ((in & ((1UL) << 22)) << (39));   \
    output |= ((in & ((1UL) << 21)) << (0));    \
    output |= ((in & ((1UL) << 20)) << (33));   \
    output |= ((in & ((1UL) << 19)) >> (6));    \
    output |= ((in & ((1UL) << 18)) << (27));   \
    output |= ((in & ((1UL) << 17)) >> (12));   \
    output |= ((in & ((1UL) << 16)) << (21));   \
    output |= ((in & ((1UL) << 15)) << (15));   \
    output |= ((in & ((1UL) << 14)) << (48));   \
    output |= ((in & ((1UL) << 13)) << (9));    \
    output |= ((in & ((1UL) << 12)) << (42));   \
    output |= ((in & ((1UL) << 11)) << (3));    \
    output |= ((in & ((1UL) << 10)) << (36));   \
    output |= ((in & ((1UL) << 9)) >> (3));     \
    output |= ((in & ((1UL) << 8)) << (30));    \
    output |= ((in & ((1UL) << 7)) << (24));    \
    output |= ((in & ((1UL) << 6)) << (57));    \
    output |= ((in & ((1UL) << 5)) << (18));    \
    output |= ((in & ((1UL) << 4)) << (51));    \
    output |= ((in & ((1UL) << 3)) << (12));    \
    output |= ((in & ((1UL) << 2)) << (45));    \
    output |= ((in & ((1UL) << 1)) << (6));     \
    output |= ((in & ((1UL) << 0)) << (39));    \
                                                \
    L = (output >> 32) & 0xFFFFFFFF;            \
    R = (output) & 0xFFFFFFFF;                  \


    

__global__ void EncryptDES(uint64_t key, uint64_t in, uint64_t expected) {
    int i, round;
    uint32_t R, L, fout; 
    uint64_t roundKey, out;

    /*
       COMPUTE_ROUND_KEY(roundKey, key)


       COMPUTE_IP(L, R, in)
     */

    /*
       for (round = 0; round < 16; round++) {
       RotateRoundKeyLeft(roundKey);
       if (round != 0 && round != 1 && round != 8 && round != 15)
       RotateRoundKeyLeft(roundKey);

       ComputeF(fout, R, roundKey);

       L ^= fout;

       Exchange_L_and_R(L, R);
       }
       Exchange_L_and_R(L, R);

       ComputeFP(out, L, R);

     */

    /*
       Logic need to be added in order to handle 
       the out == expected situation.
     */

}

void EncryptDES_host(uint64_t key, uint64_t in, uint64_t expected) {
    int i = 0, round = 0;
    uint32_t R = 0, L = 0, fout = 0; 
    uint64_t roundKey = 0UL, out;
    
    printf("sizeof(unsigned long long) is %d\n", sizeof(unsigned long long));

    COMPUTE_ROUND_KEY(roundKey, key)
    
    printf("roundKey is: \n");
    print_bits_array(roundKey);


    COMPUTE_IP(L, R, in)
    
    printf("after IP is: \n");
    printf("\t L:\n");
    print_bits_array(L);
    printf("\t R:\n");
    print_bits_array(R);
}


int main(int argc, char **argv) {

    uint64_t random_o = 0xF77D7F53F77D7F53;
    uint64_t random_k = 0x2FEABF912FEABF;

    printf("original is : \n");
    print_bits_array(random_o);
    printf("key is :\n");
    print_bits_array(random_k);
    EncryptDES_host(random_k, random_o, 0);


    return 0;
}

