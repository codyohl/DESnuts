/* This program is aimed to do the cuda implementation of DESnuts */

#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdint.h>
#include <inttypes.h>
#include <stdio.h>
#include <inttypes.h>

/**********************************************************************/
/*                                                                    */
/*                            DES TABLES                              */
/*                                                                    */
/**********************************************************************/

#define MAX_THREADS_1D 32
#define MAX_BLOCKS_1D 256
#define CONSTANT_SIZE (sizeof(int) * 8 * 64)
__constant__ int S_TABLE[CONSTANT_SIZE];


/*
 *  S Tables: Introduce nonlinearity and avalanche
 */
static int table_DES_S[8][64] = {
    /* table S[0] */
    {   13,  1,  2, 15,  8, 13,  4,  8,  6, 10, 15,  3, 11,  7,  1,  4,
        10, 12,  9,  5,  3,  6, 14, 11,  5,  0,  0, 14, 12,  9,  7,  2,
        7,  2, 11,  1,  4, 14,  1,  7,  9,  4, 12, 10, 14,  8,  2, 13,
        0, 15,  6, 12, 10,  9, 13,  0, 15,  3,  3,  5,  5,  6,  8, 11  },
    /* table S[1] */
    {    4, 13, 11,  0,  2, 11, 14,  7, 15,  4,  0,  9,  8,  1, 13, 10,
        3, 14, 12,  3,  9,  5,  7, 12,  5,  2, 10, 15,  6,  8,  1,  6,
        1,  6,  4, 11, 11, 13, 13,  8, 12,  1,  3,  4,  7, 10, 14,  7,
        10,  9, 15,  5,  6,  0,  8, 15,  0, 14,  5,  2,  9,  3,  2, 12  },
    /* table S[2] */
    {   12, 10,  1, 15, 10,  4, 15,  2,  9,  7,  2, 12,  6,  9,  8,  5,
        0,  6, 13,  1,  3, 13,  4, 14, 14,  0,  7, 11,  5,  3, 11,  8,
        9,  4, 14,  3, 15,  2,  5, 12,  2,  9,  8,  5, 12, 15,  3, 10,
        7, 11,  0, 14,  4,  1, 10,  7,  1,  6, 13,  0, 11,  8,  6, 13  },
    /* table S[3] */
    {    2, 14, 12, 11,  4,  2,  1, 12,  7,  4, 10,  7, 11, 13,  6,  1,
        8,  5,  5,  0,  3, 15, 15, 10, 13,  3,  0,  9, 14,  8,  9,  6,
        4, 11,  2,  8,  1, 12, 11,  7, 10,  1, 13, 14,  7,  2,  8, 13,
        15,  6,  9, 15, 12,  0,  5,  9,  6, 10,  3,  4,  0,  5, 14,  3  },
    /* table S[4] */
    {    7, 13, 13,  8, 14, 11,  3,  5,  0,  6,  6, 15,  9,  0, 10,  3,
        1,  4,  2,  7,  8,  2,  5, 12, 11,  1, 12, 10,  4, 14, 15,  9,
        10,  3,  6, 15,  9,  0,  0,  6, 12, 10, 11,  1,  7, 13, 13,  8,
        15,  9,  1,  4,  3,  5, 14, 11,  5, 12,  2,  7,  8,  2,  4, 14  },
    /* table S[5] */
    {   10, 13,  0,  7,  9,  0, 14,  9,  6,  3,  3,  4, 15,  6,  5, 10,
        1,  2, 13,  8, 12,  5,  7, 14, 11, 12,  4, 11,  2, 15,  8,  1,
        13,  1,  6, 10,  4, 13,  9,  0,  8,  6, 15,  9,  3,  8,  0,  7,
        11,  4,  1, 15,  2, 14, 12,  3,  5, 11, 10,  5, 14,  2,  7, 12  },
    /* table S[6] */
    {   15,  3,  1, 13,  8,  4, 14,  7,  6, 15, 11,  2,  3,  8,  4, 14,
        9, 12,  7,  0,  2,  1, 13, 10, 12,  6,  0,  9,  5, 11, 10,  5,
        0, 13, 14,  8,  7, 10, 11,  1, 10,  3,  4, 15, 13,  4,  1,  2,
        5, 11,  8,  6, 12,  7,  6, 12,  9,  0,  3,  5,  2, 14, 15,  9  },
    /* table S[7] */
    {   14,  0,  4, 15, 13,  7,  1,  4,  2, 14, 15,  2, 11, 13,  8,  1,
        3, 10, 10,  6,  6, 12, 12, 11,  5,  9,  9,  5,  0,  3,  7,  8,
        4, 15,  1, 12, 14,  8,  8,  2, 13,  4,  6,  9,  2,  1, 11,  7,
        15,  5, 12, 11,  9,  3,  7, 14,  3, 10, 10,  0,  5,  6,  0, 13  }
};

/*
void print_bits_array(uint64_t n) {
    printf("%lX\n", n);
}

void print_bits(uint64_t n) {
    for (int i = 0 ; i < 64; i++) {
        if (i == 32)
            printf("\n");
        printf("%d", ((n) & 0x8000000000000000) >> 63);
        n <<= 1;
    }
    printf("\n");
    printf("\n");
}
*/
        
#define MASK56(n) ((n) & 0x00FFFFFFFFFFFFFF)
#define MASK48(n) ((n) & 0x0000FFFFFFFFFFFF)


#define COMPUTE_ROUND_KEY(roundKey, key)        \
    roundKey |= ((key & ((1UL) << 0)) << (27));     \
    roundKey |= ((key & ((1UL) << 1)) << (18));     \
    roundKey |= ((key & ((1UL) << 2)) << (9));  \
    roundKey |= ((key & ((1UL) << 3)) << (28));     \
    roundKey |= ((key & ((1UL) << 4)) << (35));     \
    roundKey |= ((key & ((1UL) << 5)) << (42));     \
    roundKey |= ((key & ((1UL) << 6)) << (49));     \
    roundKey |= ((key & ((1UL) << 7)) << (19));     \
    roundKey |= ((key & ((1UL) << 8)) << (10));     \
    roundKey |= ((key & ((1UL) << 9)) << (1));  \
    roundKey |= ((key & ((1UL) << 10)) << (20));    \
    roundKey |= ((key & ((1UL) << 11)) << (27));    \
    roundKey |= ((key & ((1UL) << 12)) << (34));    \
    roundKey |= ((key & ((1UL) << 13)) << (41));    \
    roundKey |= ((key & ((1UL) << 14)) << (11));    \
    roundKey |= ((key & ((1UL) << 15)) << (2));     \
    roundKey |= ((key & ((1UL) << 16)) >> (7));     \
    roundKey |= ((key & ((1UL) << 17)) << (12));    \
    roundKey |= ((key & ((1UL) << 18)) << (19));    \
    roundKey |= ((key & ((1UL) << 19)) << (26));    \
    roundKey |= ((key & ((1UL) << 20)) << (33));    \
    roundKey |= ((key & ((1UL) << 21)) << (3));     \
    roundKey |= ((key & ((1UL) << 22)) >> (6));     \
    roundKey |= ((key & ((1UL) << 23)) >> (15));    \
    roundKey |= ((key & ((1UL) << 24)) << (4));     \
    roundKey |= ((key & ((1UL) << 25)) << (11));    \
    roundKey |= ((key & ((1UL) << 26)) << (18));    \
    roundKey |= ((key & ((1UL) << 27)) << (25));    \
    roundKey |= ((key & ((1UL) << 28)) >> (5));     \
    roundKey |= ((key & ((1UL) << 29)) >> (14));    \
    roundKey |= ((key & ((1UL) << 30)) >> (23));    \
    roundKey |= ((key & ((1UL) << 31)) >> (28));    \
    roundKey |= ((key & ((1UL) << 32)) << (3));     \
    roundKey |= ((key & ((1UL) << 33)) << (10));    \
    roundKey |= ((key & ((1UL) << 34)) << (17));    \
    roundKey |= ((key & ((1UL) << 35)) >> (13));    \
    roundKey |= ((key & ((1UL) << 36)) >> (22));    \
    roundKey |= ((key & ((1UL) << 37)) >> (31));    \
    roundKey |= ((key & ((1UL) << 38)) >> (36));    \
    roundKey |= ((key & ((1UL) << 39)) >> (5));     \
    roundKey |= ((key & ((1UL) << 40)) << (2));     \
    roundKey |= ((key & ((1UL) << 41)) << (9));     \
    roundKey |= ((key & ((1UL) << 42)) >> (21));    \
    roundKey |= ((key & ((1UL) << 43)) >> (30));    \
    roundKey |= ((key & ((1UL) << 44)) >> (39));    \
    roundKey |= ((key & ((1UL) << 45)) >> (44));    \
    roundKey |= ((key & ((1UL) << 46)) >> (13));    \
    roundKey |= ((key & ((1UL) << 47)) >> (6));     \
    roundKey |= ((key & ((1UL) << 48)) << (1));     \
    roundKey |= ((key & ((1UL) << 49)) >> (29));    \
    roundKey |= ((key & ((1UL) << 50)) >> (38));    \
    roundKey |= ((key & ((1UL) << 51)) >> (47));    \
    roundKey |= ((key & ((1UL) << 52)) >> (52));    \
    roundKey |= ((key & ((1UL) << 53)) >> (21));    \
    roundKey |= ((key & ((1UL) << 54)) >> (14));    \
    roundKey |= ((key & ((1UL) << 55)) >> (7));     \


#define COMPUTE_IP(L, R, in)            \
    temp = 0UL;                    \
    temp |= ((in & ((1UL) << 63)) >> (39));  \
    temp |= ((in & ((1UL) << 62)) >> (6));  \
    temp |= ((in & ((1UL) << 61)) >> (45));     \
    temp |= ((in & ((1UL) << 60)) >> (12));     \
    temp |= ((in & ((1UL) << 59)) >> (51));     \
    temp |= ((in & ((1UL) << 58)) >> (18));     \
    temp |= ((in & ((1UL) << 57)) >> (57));     \
    temp |= ((in & ((1UL) << 56)) >> (24));     \
    temp |= ((in & ((1UL) << 55)) >> (30));     \
    temp |= ((in & ((1UL) << 54)) << (3));  \
    temp |= ((in & ((1UL) << 53)) >> (36));     \
    temp |= ((in & ((1UL) << 52)) >> (3));  \
    temp |= ((in & ((1UL) << 51)) >> (42));     \
    temp |= ((in & ((1UL) << 50)) >> (9));  \
    temp |= ((in & ((1UL) << 49)) >> (48));     \
    temp |= ((in & ((1UL) << 48)) >> (15));     \
    temp |= ((in & ((1UL) << 47)) >> (21));     \
    temp |= ((in & ((1UL) << 46)) << (12));     \
    temp |= ((in & ((1UL) << 45)) >> (27));     \
    temp |= ((in & ((1UL) << 44)) << (6));  \
    temp |= ((in & ((1UL) << 43)) >> (33));     \
    temp |= ((in & ((1UL) << 42)) << (0));  \
    temp |= ((in & ((1UL) << 41)) >> (39));     \
    temp |= ((in & ((1UL) << 40)) >> (6));  \
    temp |= ((in & ((1UL) << 39)) >> (12));     \
    temp |= ((in & ((1UL) << 38)) << (21));     \
    temp |= ((in & ((1UL) << 37)) >> (18));     \
    temp |= ((in & ((1UL) << 36)) << (15));     \
    temp |= ((in & ((1UL) << 35)) >> (24));     \
    temp |= ((in & ((1UL) << 34)) << (9));  \
    temp |= ((in & ((1UL) << 33)) >> (30));     \
    temp |= ((in & ((1UL) << 32)) << (3));  \
    temp |= ((in & ((1UL) << 31)) >> (3));  \
    temp |= ((in & ((1UL) << 30)) << (30));     \
    temp |= ((in & ((1UL) << 29)) >> (9));  \
    temp |= ((in & ((1UL) << 28)) << (24));     \
    temp |= ((in & ((1UL) << 27)) >> (15));     \
    temp |= ((in & ((1UL) << 26)) << (18));     \
    temp |= ((in & ((1UL) << 25)) >> (21));     \
    temp |= ((in & ((1UL) << 24)) << (12));     \
    temp |= ((in & ((1UL) << 23)) << (6));  \
    temp |= ((in & ((1UL) << 22)) << (39));     \
    temp |= ((in & ((1UL) << 21)) << (0));  \
    temp |= ((in & ((1UL) << 20)) << (33));     \
    temp |= ((in & ((1UL) << 19)) >> (6));  \
    temp |= ((in & ((1UL) << 18)) << (27));     \
    temp |= ((in & ((1UL) << 17)) >> (12));     \
    temp |= ((in & ((1UL) << 16)) << (21));     \
    temp |= ((in & ((1UL) << 15)) << (15));     \
    temp |= ((in & ((1UL) << 14)) << (48));     \
    temp |= ((in & ((1UL) << 13)) << (9));  \
    temp |= ((in & ((1UL) << 12)) << (42));     \
    temp |= ((in & ((1UL) << 11)) << (3));  \
    temp |= ((in & ((1UL) << 10)) << (36));     \
    temp |= ((in & ((1UL) << 9)) >> (3));   \
    temp |= ((in & ((1UL) << 8)) << (30));  \
    temp |= ((in & ((1UL) << 7)) << (24));  \
    temp |= ((in & ((1UL) << 6)) << (57));  \
    temp |= ((in & ((1UL) << 5)) << (18));  \
    temp |= ((in & ((1UL) << 4)) << (51));  \
    temp |= ((in & ((1UL) << 3)) << (12));  \
    temp |= ((in & ((1UL) << 2)) << (45));  \
    temp |= ((in & ((1UL) << 1)) << (6));   \
    temp |= ((in & ((1UL) << 0)) << (39));  \
    L = (temp >> 32) & 0xFFFFFFFF;            \
    R = (temp) & 0xFFFFFFFF;                  \


#define COMPUTE_FP(out, L, R)                   \
    temp = L;                                   \
    temp = (temp << 32) | R;                        \
    out |= ((temp & ((1UL) << 63)) >> (57));    \
    out |= ((temp & ((1UL) << 62)) >> (48));    \
    out |= ((temp & ((1UL) << 61)) >> (39));    \
    out |= ((temp & ((1UL) << 60)) >> (30));    \
    out |= ((temp & ((1UL) << 59)) >> (21));    \
    out |= ((temp & ((1UL) << 58)) >> (12));    \
    out |= ((temp & ((1UL) << 57)) >> (3));     \
    out |= ((temp & ((1UL) << 56)) << (6));     \
    out |= ((temp & ((1UL) << 55)) >> (51));    \
    out |= ((temp & ((1UL) << 54)) >> (42));    \
    out |= ((temp & ((1UL) << 53)) >> (33));    \
    out |= ((temp & ((1UL) << 52)) >> (24));    \
    out |= ((temp & ((1UL) << 51)) >> (15));    \
    out |= ((temp & ((1UL) << 50)) >> (6));     \
    out |= ((temp & ((1UL) << 49)) << (3));     \
    out |= ((temp & ((1UL) << 48)) << (12));    \
    out |= ((temp & ((1UL) << 47)) >> (45));    \
    out |= ((temp & ((1UL) << 46)) >> (36));    \
    out |= ((temp & ((1UL) << 45)) >> (27));    \
    out |= ((temp & ((1UL) << 44)) >> (18));    \
    out |= ((temp & ((1UL) << 43)) >> (9));     \
    out |= ((temp & ((1UL) << 42)) << (0));     \
    out |= ((temp & ((1UL) << 41)) << (9));     \
    out |= ((temp & ((1UL) << 40)) << (18));    \
    out |= ((temp & ((1UL) << 39)) >> (39));    \
    out |= ((temp & ((1UL) << 38)) >> (30));    \
    out |= ((temp & ((1UL) << 37)) >> (21));    \
    out |= ((temp & ((1UL) << 36)) >> (12));    \
    out |= ((temp & ((1UL) << 35)) >> (3));     \
    out |= ((temp & ((1UL) << 34)) << (6));     \
    out |= ((temp & ((1UL) << 33)) << (15));    \
    out |= ((temp & ((1UL) << 32)) << (24));    \
    out |= ((temp & ((1UL) << 31)) >> (24));    \
    out |= ((temp & ((1UL) << 30)) >> (15));    \
    out |= ((temp & ((1UL) << 29)) >> (6));     \
    out |= ((temp & ((1UL) << 28)) << (3));     \
    out |= ((temp & ((1UL) << 27)) << (12));    \
    out |= ((temp & ((1UL) << 26)) << (21));    \
    out |= ((temp & ((1UL) << 25)) << (30));    \
    out |= ((temp & ((1UL) << 24)) << (39));    \
    out |= ((temp & ((1UL) << 23)) >> (18));    \
    out |= ((temp & ((1UL) << 22)) >> (9));     \
    out |= ((temp & ((1UL) << 21)) << (0));     \
    out |= ((temp & ((1UL) << 20)) << (9));     \
    out |= ((temp & ((1UL) << 19)) << (18));    \
    out |= ((temp & ((1UL) << 18)) << (27));    \
    out |= ((temp & ((1UL) << 17)) << (36));    \
    out |= ((temp & ((1UL) << 16)) << (45));    \
    out |= ((temp & ((1UL) << 15)) >> (12));    \
    out |= ((temp & ((1UL) << 14)) >> (3));     \
    out |= ((temp & ((1UL) << 13)) << (6));     \
    out |= ((temp & ((1UL) << 12)) << (15));    \
    out |= ((temp & ((1UL) << 11)) << (24));    \
    out |= ((temp & ((1UL) << 10)) << (33));    \
    out |= ((temp & ((1UL) << 9)) << (42));     \
    out |= ((temp & ((1UL) << 8)) << (51));     \
    out |= ((temp & ((1UL) << 7)) >> (6));  \
    out |= ((temp & ((1UL) << 6)) << (3));  \
    out |= ((temp & ((1UL) << 5)) << (12));     \
    out |= ((temp & ((1UL) << 4)) << (21));     \
    out |= ((temp & ((1UL) << 3)) << (30));     \
    out |= ((temp & ((1UL) << 2)) << (39));     \
    out |= ((temp & ((1UL) << 1)) << (48));     \
    out |= ((temp & ((1UL) << 0)) << (57));     \

#define COMPUTE_P(out, in)  \
    out |= ((in & ((1UL) << 0)) << (11));   \
    out |= ((in & ((1UL) << 1)) << (16));   \
    out |= ((in & ((1UL) << 2)) << (3));    \
    out |= ((in & ((1UL) << 3)) << (24));   \
    out |= ((in & ((1UL) << 4)) << (21));   \
    out |= ((in & ((1UL) << 5)) << (5));    \
    out |= ((in & ((1UL) << 6)) << (14));   \
    out |= ((in & ((1UL) << 7)) >> (7));    \
    out |= ((in & ((1UL) << 8)) << (5));    \
    out |= ((in & ((1UL) << 9)) << (12));   \
    out |= ((in & ((1UL) << 10)) >> (7));   \
    out |= ((in & ((1UL) << 11)) << (17));  \
    out |= ((in & ((1UL) << 12)) << (17));  \
    out |= ((in & ((1UL) << 13)) >> (6));   \
    out |= ((in & ((1UL) << 14)) << (4));   \
    out |= ((in & ((1UL) << 15)) << (9));   \
    out |= ((in & ((1UL) << 16)) << (15));  \
    out |= ((in & ((1UL) << 17)) << (5));   \
    out |= ((in & ((1UL) << 18)) >> (6));   \
    out |= ((in & ((1UL) << 19)) >> (13));  \
    out |= ((in & ((1UL) << 20)) << (6));   \
    out |= ((in & ((1UL) << 21)) >> (19));  \
    out |= ((in & ((1UL) << 22)) >> (6));   \
    out |= ((in & ((1UL) << 23)) >> (15));  \
    out |= ((in & ((1UL) << 24)) >> (10));  \
    out |= ((in & ((1UL) << 25)) << (5));   \
    out |= ((in & ((1UL) << 26)) >> (22));  \
    out |= ((in & ((1UL) << 27)) >> (8));   \
    out |= ((in & ((1UL) << 28)) >> (27));  \
    out |= ((in & ((1UL) << 29)) >> (20));  \
    out |= ((in & ((1UL) << 30)) >> (15));  \
    out |= ((in & ((1UL) << 31)) >> (8));   \


#define COMPUTE_EXPANSION_E(expB, Rin)        \
    expB |= ((R & ((1UL) << 31)) >> (31));  \
    expB |= ((R & ((1UL) << 0)) << (1));    \
    expB |= ((R & ((1UL) << 1)) << (1));    \
    expB |= ((R & ((1UL) << 2)) << (1));    \
    expB |= ((R & ((1UL) << 3)) << (1));    \
    expB |= ((R & ((1UL) << 4)) << (1));    \
    expB |= ((R & ((1UL) << 3)) << (3));    \
    expB |= ((R & ((1UL) << 4)) << (3));    \
    expB |= ((R & ((1UL) << 5)) << (3));    \
    expB |= ((R & ((1UL) << 6)) << (3));    \
    expB |= ((R & ((1UL) << 7)) << (3));    \
    expB |= ((R & ((1UL) << 8)) << (3));    \
    expB |= ((R & ((1UL) << 7)) << (5));    \
    expB |= ((R & ((1UL) << 8)) << (5));    \
    expB |= ((R & ((1UL) << 9)) << (5));    \
    expB |= ((R & ((1UL) << 10)) << (5));   \
    expB |= ((R & ((1UL) << 11)) << (5));   \
    expB |= ((R & ((1UL) << 12)) << (5));   \
    expB |= ((R & ((1UL) << 11)) << (7));   \
    expB |= ((R & ((1UL) << 12)) << (7));   \
    expB |= ((R & ((1UL) << 13)) << (7));   \
    expB |= ((R & ((1UL) << 14)) << (7));   \
    expB |= ((R & ((1UL) << 15)) << (7));   \
    expB |= ((R & ((1UL) << 16)) << (7));   \
    expB |= ((R & ((1UL) << 15)) << (9));   \
    expB |= ((R & ((1UL) << 16)) << (9));   \
    expB |= ((R & ((1UL) << 17)) << (9));   \
    expB |= ((R & ((1UL) << 18)) << (9));   \
    expB |= ((R & ((1UL) << 19)) << (9));   \
    expB |= ((R & ((1UL) << 20)) << (9));   \
    expB |= ((R & ((1UL) << 19)) << (11));  \
    expB |= ((R & ((1UL) << 20)) << (11));  \
    expB |= ((R & ((1UL) << 21)) << (11));  \
    expB |= ((R & ((1UL) << 22)) << (11));  \
    expB |= ((R & ((1UL) << 23)) << (11));  \
    expB |= ((R & ((1UL) << 24)) << (11));  \
    expB |= ((R & ((1UL) << 23)) << (13));  \
    expB |= ((R & ((1UL) << 24)) << (13));  \
    expB |= ((R & ((1UL) << 25)) << (13));  \
    expB |= ((R & ((1UL) << 26)) << (13));  \
    expB |= ((R & ((1UL) << 27)) << (13));  \
    expB |= ((R & ((1UL) << 28)) << (13));  \
    expB |= ((R & ((1UL) << 27)) << (15));  \
    expB |= ((R & ((1UL) << 28)) << (15));  \
    expB |= ((R & ((1UL) << 29)) << (15));  \
    expB |= ((R & ((1UL) << 30)) << (15));  \
    expB |= ((R & ((1UL) << 31)) << (15));  \
    expB |= ((R & ((1UL) << 0)) << (47));   \


#define COMPUTE_PC2(subkey, roundKey)       \
    subkey |= ((roundKey & ((1UL) << 24)) >> (24));     \
    subkey |= ((roundKey & ((1UL) << 27)) >> (26));     \
    subkey |= ((roundKey & ((1UL) << 20)) >> (18));     \
    subkey |= ((roundKey & ((1UL) << 6)) >> (3));   \
    subkey |= ((roundKey & ((1UL) << 14)) >> (10));     \
    subkey |= ((roundKey & ((1UL) << 10)) >> (5));  \
    subkey |= ((roundKey & ((1UL) << 3)) << (3));   \
    subkey |= ((roundKey & ((1UL) << 22)) >> (15));     \
    subkey |= ((roundKey & ((1UL) << 0)) << (8));   \
    subkey |= ((roundKey & ((1UL) << 17)) >> (8));  \
    subkey |= ((roundKey & ((1UL) << 7)) << (3));   \
    subkey |= ((roundKey & ((1UL) << 12)) >> (1));  \
    subkey |= ((roundKey & ((1UL) << 8)) << (4));   \
    subkey |= ((roundKey & ((1UL) << 23)) >> (10));     \
    subkey |= ((roundKey & ((1UL) << 11)) << (3));  \
    subkey |= ((roundKey & ((1UL) << 5)) << (10));  \
    subkey |= ((roundKey & ((1UL) << 16)) >> (0));  \
    subkey |= ((roundKey & ((1UL) << 26)) >> (9));  \
    subkey |= ((roundKey & ((1UL) << 1)) << (17));  \
    subkey |= ((roundKey & ((1UL) << 9)) << (10));  \
    subkey |= ((roundKey & ((1UL) << 19)) << (1));  \
    subkey |= ((roundKey & ((1UL) << 25)) >> (4));  \
    subkey |= ((roundKey & ((1UL) << 4)) << (18));  \
    subkey |= ((roundKey & ((1UL) << 15)) << (8));  \
    subkey |= ((roundKey & ((1UL) << 54)) >> (30));     \
    subkey |= ((roundKey & ((1UL) << 43)) >> (18));     \
    subkey |= ((roundKey & ((1UL) << 36)) >> (10));     \
    subkey |= ((roundKey & ((1UL) << 29)) >> (2));  \
    subkey |= ((roundKey & ((1UL) << 49)) >> (21));     \
    subkey |= ((roundKey & ((1UL) << 40)) >> (11));     \
    subkey |= ((roundKey & ((1UL) << 48)) >> (18));     \
    subkey |= ((roundKey & ((1UL) << 30)) << (1));  \
    subkey |= ((roundKey & ((1UL) << 52)) >> (20));     \
    subkey |= ((roundKey & ((1UL) << 44)) >> (11));     \
    subkey |= ((roundKey & ((1UL) << 37)) >> (3));  \
    subkey |= ((roundKey & ((1UL) << 33)) << (2));  \
    subkey |= ((roundKey & ((1UL) << 46)) >> (10));     \
    subkey |= ((roundKey & ((1UL) << 35)) << (2));  \
    subkey |= ((roundKey & ((1UL) << 50)) >> (12));     \
    subkey |= ((roundKey & ((1UL) << 41)) >> (2));  \
    subkey |= ((roundKey & ((1UL) << 28)) << (12));     \
    subkey |= ((roundKey & ((1UL) << 53)) >> (12));     \
    subkey |= ((roundKey & ((1UL) << 51)) >> (9));  \
    subkey |= ((roundKey & ((1UL) << 55)) >> (12));     \
    subkey |= ((roundKey & ((1UL) << 32)) << (12));     \
    subkey |= ((roundKey & ((1UL) << 45)) >> (0));  \
    subkey |= ((roundKey & ((1UL) << 39)) << (7));  \
    subkey |= ((roundKey & ((1UL) << 42)) << (5));  \


#define COMPUTES_LOOKUP(k, sout, expandedBlock)     \
    sout |= S_TABLE[k * 64 + ((expandedBlock >> (6 * k)) & 0x3F)] << (4 * k);      \

/* This is the host code
#define COMPUTES_LOOKUP(k, sout, expandedBlock)     \
    sout |= table_DES_S[k][(expandedBlock >> (6 * k)) & 0x3F] << (4 * k);      \
*/

/*
uint32_t COMPUTE_F(uint32_t fout, uint32_t R, uint64_t roundKey) {
    uint64_t expandedBlock = 0UL, subkey = 0UL;
    uint32_t sout = 0;
    int i, k;

    COMPUTE_EXPANSION_E(expandedBlock, R)

    printf("expanded E is : \n");
    print_bits_array(expandedBlock);

    COMPUTE_PC2(subkey, roundKey)

    printf("subkey is :\n");
    print_bits_array(subkey);

    expandedBlock ^= subkey;
    // Mask expandedBlock
    expandedBlock = MASK48(expandedBlock);
    printf("Expanded E is :\n");
    print_bits_array(expandedBlock);

    for (k = 0; k < 8; k++) {
        COMPUTES_LOOKUP(k, sout, expandedBlock)

        printf("sout @ %d is :\n", k);
        print_bits_array(sout);
    }

    COMPUTE_P(fout, sout)

    printf("fout is :\n");
    print_bits_array(fout);
    printf("sout is :\n");
    print_bits_array(sout);

    return fout;
}
*/


#define ROTATE_ROUND_KEY_LEFT(roundK)         \
    uint64_t bit27 = ((roundK & ((1UL) << 27)) >> 27);\
    uint64_t bit55 = ((roundK & ((1UL) << 55)) >> 27);\
    roundK <<= 1;                             \
    temp = roundK & 0x00FFFFFFEFFFFFFE;            \
    roundK = temp | bit27 | bit55;           \




#define EXCHANGE_L_AND_R(L, R)                  \
    temp = L;                                   \
    L = R;                                      \
    R = temp;                                   \
    

__global__ void EncryptDES_device(uint64_t in, uint64_t expected, uint64_t* result, uint64_t bound) {

    int blockId = blockIdx.x + blockIdx.y * gridDim.x; 
    int threadId = blockId * (blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x) + threadIdx.x;
    uint64_t key = threadId * bound;
    uint64_t counter = 0;
    
    while (counter != bound) {

        uint32_t R = 0, L = 0; 
        uint64_t roundKey = 0UL, out = 0UL, temp = 0UL;

        COMPUTE_ROUND_KEY(roundKey, key)

        COMPUTE_IP(L, R, in)

        for (int round = 0; round < 16; round++) {
            uint64_t expandedBlock = 0UL, subkey = 0UL;
            uint32_t sout = 0;
            uint32_t fout = 0;

            ROTATE_ROUND_KEY_LEFT(roundKey)

            if (round != 0 && round != 1 && round != 8 && round != 15) {
                ROTATE_ROUND_KEY_LEFT(roundKey)
            }


            COMPUTE_EXPANSION_E(expandedBlock, R)

            COMPUTE_PC2(subkey, roundKey)


            expandedBlock ^= subkey;
            expandedBlock = MASK48(expandedBlock);

            for (int i = 0; i < 8; i++) {
                COMPUTES_LOOKUP(i, sout, expandedBlock)
            }

            COMPUTE_P(fout, sout)

                L ^= fout;

            EXCHANGE_L_AND_R(L, R)

        }
        EXCHANGE_L_AND_R(L, R)

        COMPUTE_FP(out, L, R)

        if (out == expected) {
            *result = out;
             // asm("trap;");
        }
        counter++;
        key++;
    }
    __syncthreads();
}

/*
void EncryptDES_host(uint64_t key, uint64_t in, uint64_t expected) {
    uint32_t R = 0, L = 0; 
    uint64_t roundKey = 0UL, out = 0UL, temp = 0UL;
    
    printf("sizeof(unsigned long long) is %d\n", sizeof(unsigned long long));

    COMPUTE_ROUND_KEY(roundKey, key)
    
    printf("roundKey is: \n");
    print_bits_array(roundKey);


    COMPUTE_IP(L, R, in)
    
    printf("after IP is: \n");
    printf("\t L:\n");
    print_bits_array(L);
    printf("\t R:\n");
    print_bits_array(R);

    for (int round = 0; round < 16; round++) {
        uint64_t expandedBlock = 0UL, subkey = 0UL;
        uint32_t sout = 0;
        uint32_t fout = 0;

        printf("------------------------- ROUND %d ----------------------\n", round);
    

        ROTATE_ROUND_KEY_LEFT(roundKey)
        
        printf("\t roundKey:\n");
        print_bits_array(roundKey);

        if (round != 0 && round != 1 && round != 8 && round != 15) {
            ROTATE_ROUND_KEY_LEFT(roundKey)
        }


        COMPUTE_EXPANSION_E(expandedBlock, R)

        printf("expanded E is : \n");
        print_bits_array(expandedBlock);

        COMPUTE_PC2(subkey, roundKey)

        printf("subkey is :\n");
        print_bits_array(subkey);

        expandedBlock ^= subkey;
        // Mask expandedBlock
        expandedBlock = MASK48(expandedBlock);
        printf("Expanded E is :\n");
        print_bits_array(expandedBlock);

        for (int i = 0; i < 8; i++) {
               Comment out for compilation of the device code

            COMPUTES_LOOKUP(i, sout, expandedBlock)
            printf("sout @ %d is :\n", i);
            print_bits_array(sout);
        }

        COMPUTE_P(fout, sout)

        printf("fout is :\n");
        print_bits_array(fout);
        printf("sout is :\n");
        print_bits_array(sout);

        printf("f is : \n");
        print_bits_array(fout);

        L ^= fout;

        printf("L^f is : \n");
        print_bits_array(L);
        
        EXCHANGE_L_AND_R(L, R)

        printf("------------------------- ROUND %d end ------------------\n", round);
         
    }
    EXCHANGE_L_AND_R(L, R)

    COMPUTE_FP(out, L, R)
    
    printf("FP out is \n");
    print_bits_array(out);

}
*/


int main(int argc, char **argv) {

    uint64_t random_o = 0xF77D7F53F77D7F53;
    // uint64_t random_k = 0x2FEABF912FEABF;
    uint64_t expected = 0xDF86B0B30BD2530A;

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float milliseconds = 0;

    uint64_t *result_host = (uint64_t *)calloc(1, sizeof(uint64_t));
    uint64_t *result_device;
    hipMalloc(&result_device, sizeof(uint64_t));


    hipMemcpy(result_device, result_host, sizeof(uint64_t),  hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(S_TABLE), table_DES_S, CONSTANT_SIZE);

    int threads = MAX_THREADS_1D / 2;
    int blocks = (MAX_BLOCKS_1D - 1);

    uint64_t overall_total = 0x0FFFFFFFFFFFFFFFULL;
    uint64_t target_total = 0xFFFFFFFFFFULL;

    dim3 dimGrid(blocks, blocks);
    dim3 dimBlock(threads, threads);

    hipEventRecord(start, 0);
    hipEventSynchronize(start);

    EncryptDES_device<<<dimGrid, dimBlock>>>(random_o, expected, result_device, (target_total / (blocks * blocks * threads * threads)));
    hipDeviceSynchronize();

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);

    printf("Cuda Execution Report: \n");
    printf("Targetting number of testing key - %" PRIu64 "\n", target_total);
    printf("Time spent %0.8f ms\n", milliseconds);
    printf("\n");
    printf("Estimated time to crack DES is %0.8f ms\n", (overall_total * 1.0 / target_total) * milliseconds);

    hipMemcpy(result_host, result_device, sizeof(uint64_t), hipMemcpyDeviceToHost);

    if (*result_host != 0x0)
        printf("Key found: %lX\n", *result_host);


    free(result_host);
    hipFree(result_device);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    return 0;
}

